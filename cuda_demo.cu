#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
  printf("hello world\n");
}

int main() {
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  
  return 0;
}
